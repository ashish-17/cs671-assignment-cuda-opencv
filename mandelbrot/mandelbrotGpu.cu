
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define TILE_WIDTH 36
#define TILE_HEIGHT 36

__device__ int calc_mandel(float c_re, float c_im, int count)
{
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < count; ++i) {

        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re*z_re - z_im*z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    return i;
}

__device__ int mandelbrot_calc(
    float x0, float y0, float x1, float y1,
    int width, int height,
    int row, int col,
    int maxIterations)
{
    float dx = (x1 - x0) / width;
    float dy = (y1 - y0) / height;
	float x = x0 + col * dx;
	float y = y0 + row * dy;
	int val = calc_mandel(x, y, maxIterations);

	return val;
}

__global__ void mandelbrot_kernel(float x0, float y0, float x1, float y1, int width, int height, int maxIterations, int* output) {
	int col = blockIdx.x*blockDim.x + threadIdx.x;	
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	if (row < height && col < width) {
		output[row*width + col] = mandelbrot_calc(x0, y0, x1, y1, width, height, row, col, maxIterations);
	}
}

void mandelbrotGpu(
    float x0, float y0, float x1, float y1,
    int width, int height,
    int maxIterations,
    int output[]) {

	int* d_output;
	float millisec = 0.0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start);
	hipMalloc(&d_output, width*height*sizeof(int));

	int nTilesX = width / TILE_WIDTH + (width % TILE_WIDTH == 0) ? 0 : 1;
	int nTilesY = height / TILE_HEIGHT + (height % TILE_HEIGHT == 0) ? 0 : 1;
	dim3 threadsPerBlock(TILE_WIDTH, TILE_HEIGHT);
	dim3 blocksPerGrid(nTilesX, nTilesY);
	mandelbrot_kernel<<<blocksPerGrid, threadsPerBlock>>>(x0, y0, x1, y1, width, height, maxIterations, d_output);
	
	hipMemcpy(output, d_output, width*height*sizeof(int), hipMemcpyDeviceToHost);		
	
	hipFree(d_output);		

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&millisec, start, stop);
	printf("\ncuda time = %f\n", millisec);
}
